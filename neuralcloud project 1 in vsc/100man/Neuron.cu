#include "hip/hip_runtime.h"
#include "Neuron.h"
#include <cmath>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// CUDA 커널 함수: 활성화 함수 (Sigmoid)
__device__ double sigmoid(double x) {
    return 1.0 / (1.0 + exp(-x));
}

// CUDA 커널 함수: 활성화 함수 미분
__device__ double sigmoid_derivative(double x) {
    return x * (1.0 - x);
}

// 뉴런 생성자: GPU 메모리 할당
Neuron::Neuron(int num_inputs) : value(0.0), bias(0.0), gradient(0.0), num_inputs(num_inputs) {
    hipMalloc((void**)&weights, num_inputs * sizeof(double));
    hipMalloc((void**)&inputs, num_inputs * sizeof(Neuron*));

    // 랜덤 가중치 초기화
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
    hiprandGenerateUniformDouble(gen, weights, num_inputs);
    hiprandDestroyGenerator(gen);

    // 편향 초기화
    bias = ((double)rand() / RAND_MAX) * 2 - 1; // -1 ~ 1 사이의 값
}

// 뉴런 소멸자: GPU 메모리 해제
Neuron::~Neuron() {
    hipFree(weights);
    hipFree(inputs);
}

// 순방향 계산 CUDA 커널
__global__ void forward_kernel(Neuron* neuron) {
    double sum = neuron->bias;
    for (int i = 0; i < neuron->num_inputs; ++i) {
        sum += neuron->inputs[i]->get_output() * neuron->weights[i];
    }
    neuron->value = sigmoid(sum);
}

// 순방향 계산 함수 호출
double Neuron::forward() {
    forward_kernel<<<1, 1>>>(this);
    hipDeviceSynchronize();
    return value;
}

// 역전파 CUDA 커널
__global__ void backward_kernel(Neuron* neuron, double target, bool is_output) {
    if (is_output) {
        neuron->gradient = (target - neuron->value) * sigmoid_derivative(neuron->value);
    } else {
        double error = 0.0;
        for (int i = 0; i < neuron->num_inputs; ++i) {
            error += neuron->inputs[i]->gradient * neuron->weights[i];
        }
        neuron->gradient = error * sigmoid_derivative(neuron->value);
    }
}

// 역전파 계산 함수 호출
void Neuron::backward(double target, bool is_output) {
    backward_kernel<<<1, 1>>>(this, target, is_output);
    hipDeviceSynchronize();
}

// 가중치 업데이트 CUDA 커널
__global__ void update_weights_kernel(Neuron* neuron, double learning_rate) {
    for (int i = 0; i < neuron->num_inputs; ++i) {
        neuron->weights[i] += learning_rate * neuron->gradient * neuron->inputs[i]->get_output();
    }
    neuron->bias += learning_rate * neuron->gradient;
}

// 가중치 업데이트 함수 호출
void Neuron::update_weights(double learning_rate) {
    update_weights_kernel<<<1, 1>>>(this, learning_rate);
    hipDeviceSynchronize();
}

// 입력 값 설정
void Neuron::set_input(double input_value) {
    value = input_value;
}

// 출력 값 반환
double Neuron::get_output() const {
    return value;
}

// 입력 뉴런 연결
void Neuron::connect_input(Neuron* neuron) {
    // GPU 메모리에 직접 입력 뉴런 연결
    hipMemcpy(&inputs[num_inputs++], &neuron, sizeof(Neuron*), hipMemcpyHostToDevice);
}
