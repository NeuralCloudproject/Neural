#include <iostream>
#include "AICore.h"
#include <random>
#include <hip/hip_runtime.h>

int main() {
    const int TOTAL_NEURONS = 100000; // 총 뉴런 수
    const int INPUT_LAYER_SIZE = 1000;
    const int OUTPUT_LAYER_SIZE = 1000;
    const int HIDDEN_LAYER_SIZE = (TOTAL_NEURONS - INPUT_LAYER_SIZE - OUTPUT_LAYER_SIZE) / 3;

    AICore ai(INPUT_LAYER_SIZE, HIDDEN_LAYER_SIZE, OUTPUT_LAYER_SIZE);

    // 랜덤 연결 설정 (최대 100,000개의 연결)
    std::default_random_engine generator;
    std::uniform_int_distribution<int> distribution(0, INPUT_LAYER_SIZE - 1);

    for (auto& neuron : ai.hidden_layer_1) {
        for (int i = 0; i < 100000; ++i) {
            neuron.connect_input(&ai.input_layer[distribution(generator)]);
        }
    }

    for (auto& neuron : ai.hidden_layer_2) {
        for (int i = 0; i < 100000; ++i) {
            neuron.connect_input(&ai.hidden_layer_1[distribution(generator)]);
        }
    }

    for (auto& neuron : ai.hidden_layer_3) {
        for (int i = 0; i < 100000; ++i) {
            neuron.connect_input(&ai.hidden_layer_2[distribution(generator)]);
        }
    }

    for (auto& neuron : ai.output_layer) {
        for (int i = 0; i < 100000; ++i) {
            neuron.connect_input(&ai.hidden_layer_3[distribution(generator)]);
        }
    }

    std::cout << "10만 개 뉴런 모델이 GPU에서 생성 및 연결되었습니다!" << std::endl;

    return 0;
}
